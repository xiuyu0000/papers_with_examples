#include "hip/hip_runtime.h"
/**
 * Copyright 2022   Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>

#include "adaptive_gridsampler_kernel.cuh"
#define BLOCK_SIZE 256

template <typename scalar_t>
__global__ void kernel_adaptive_gridsampler_update_output(
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> img,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> kernels,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> offsets_h,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> offsets_v,
    const int *ptr_offset_unit,
    const int *ptr_padding,
    at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> output,
    const size_t n) {
    auto global_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (global_idx >= n)
        return;

    auto dim_b = output.size(0);
    auto dim_c = output.size(1);
    auto dim_h = output.size(2);
    auto dim_w = output.size(3);

    auto idb = (global_idx / (dim_c * dim_h * dim_w)) % dim_b;
    auto idc = (global_idx / (dim_h * dim_w)) % dim_c;
    auto idy = (global_idx / dim_w) % dim_h;
    auto idx = global_idx % dim_w;

    if (idx >= dim_w || idy >= dim_h)
        return;

    int offset_unit = *ptr_offset_unit;
    int padding = *ptr_padding;

    int k_size = sqrt(static_cast<float>(kernels.size(1)));
    float w = static_cast<float>(img.size(3) - 2 * padding);
    float h = static_cast<float>(img.size(2) - 2 * padding);

    scalar_t result = 0;
    for (int k_y = 0; k_y < k_size; ++k_y) {
        for (int k_x = 0; k_x < k_size; ++k_x) {
            scalar_t offset_h = offsets_h[idb][k_size * k_y + k_x][idy][idx] * offset_unit;
            scalar_t offset_v = offsets_v[idb][k_size * k_y + k_x][idy][idx] * offset_unit;

            scalar_t p_x = static_cast<scalar_t>(idx + 0.5) / dim_w * w + k_x + offset_h - 0.5;
            scalar_t p_y = static_cast<scalar_t>(idy + 0.5) / dim_h * h + k_y + offset_v - 0.5;
            scalar_t alpha = p_x - floor(p_x);
            scalar_t beta = p_y - floor(p_y);

            int xL = max(min(static_cast<int>(floor(p_x)), static_cast<int>(w + 2 * padding - 1)), 0);
            int xR = max(min(xL + 1, static_cast<int>(w + 2 * padding - 1)), 0);
            int yT = max(min(static_cast<int>(floor(p_y)), static_cast<int>(h + 2 * padding - 1)), 0);
            int yB = max(min(yT + 1, static_cast<int>(h + 2 * padding - 1)), 0);

            scalar_t val = 0;
            val += (1 - alpha) * (1 - beta) * img[idb][idc][yT][xL];
            val += alpha * (1 - beta) * img[idb][idc][yT][xR];
            val += (1 - alpha) * beta * img[idb][idc][yB][xL];
            val += alpha * beta * img[idb][idc][yB][xR];

            result += val * kernels[idb][k_size * k_y + k_x][idy][idx];
        }
    }
    output[idb][idc][idy][idx] = result;
}

void adaptive_gridsampler_kernel_forward(
    const at::Tensor &img,
    const at::Tensor &kernels,
    const at::Tensor &offsets_h,
    const at::Tensor &offsets_v,
    const int *offset_unit,
    const int *padding,
    at::Tensor *output) {

    kernel_adaptive_gridsampler_update_output<float><<<((*output).numel() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        img.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        kernels.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offsets_h.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offsets_v.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offset_unit,
        padding,
        (*output).packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        (*output).numel());

    checkCudaErrors(hipGetLastError());
}

template <typename scalar_t>
__global__ void kernel_adaptive_gridsampler_backward(
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> img,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> kernels,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> offsets_h,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> offsets_v,
    const int *ptr_offset_unit,
    const int *ptr_padding,
    const at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> gradOutput,
    at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> gradInput_kernels,
    at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> gradInput_offsets_h,
    at::PackedTensorAccessor32<scalar_t, 4, at::RestrictPtrTraits> gradInput_offsets_v,
    const size_t n) {
    auto global_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (global_idx >= n)
        return;

    auto dim_b = gradInput_kernels.size(0);
    auto dim_c = gradInput_kernels.size(1);
    auto dim_h = gradInput_kernels.size(2);
    auto dim_w = gradInput_kernels.size(3);

    auto idb = (global_idx / (dim_c * dim_h * dim_w)) % dim_b;
    auto idc = (global_idx / (dim_h * dim_w)) % dim_c;
    auto idy = (global_idx / dim_w) % dim_h;
    auto idx = global_idx % dim_w;

    if (idx >= dim_w || idx >= dim_h)
        return;

    int k_size = sqrt(static_cast<float>(dim_c));
    int k_y = idc / k_size;
    int k_x = idc % k_size;

    int offset_unit = *ptr_offset_unit;
    int padding = *ptr_padding;

    scalar_t offset_h = offsets_h[idb][idc][idy][idx] * offset_unit;
    scalar_t offset_v = offsets_v[idb][idc][idy][idx] * offset_unit;

    float w = static_cast<float>(img.size(3) - 2 * padding);
    float h = static_cast<float>(img.size(2) - 2 * padding);

    scalar_t p_x = static_cast<scalar_t>(idx + 0.5) / dim_w * w + k_x + offset_h - 0.5;
    scalar_t p_y = static_cast<scalar_t>(idy + 0.5) / dim_h * h + k_y + offset_v - 0.5;
    scalar_t alpha = p_x - floor(p_x);
    scalar_t beta = p_y - floor(p_y);

    int xL = max(min(static_cast<int>(floor(p_x)), static_cast<int>(w + 2 * padding - 1)), 0);
    int xR = max(min(xL + 1, static_cast<int>(w + 2 * padding - 1)), 0);
    int yT = max(min(static_cast<int>(floor(p_y)), static_cast<int>(h + 2 * padding - 1)), 0);
    int yB = max(min(yT + 1, static_cast<int>(h + 2 * padding - 1)), 0);

    scalar_t grad_kernels = 0;
    scalar_t grad_offset_h = 0;
    scalar_t grad_offset_v = 0;
    for (int c = 0; c < img.size(1); ++c) {
        scalar_t c_tl = img[idb][c][yT][xL];
        scalar_t c_tr = img[idb][c][yT][xR];
        scalar_t c_bl = img[idb][c][yB][xL];
        scalar_t c_br = img[idb][c][yB][xR];

        scalar_t grad = 0;
        grad += (1 - alpha) * (1 - beta) * c_tl;
        grad += alpha * (1 - beta) * c_tr;
        grad += (1 - alpha) * beta * c_bl;
        grad += alpha * beta * c_br;
        grad_kernels += grad * gradOutput[idb][c][idy][idx];

        grad = (beta - 1) * c_tl + (1 - beta) * c_tr - beta * c_bl + beta * c_br;
        grad_offset_h += kernels[idb][idc][idy][idx] * grad * gradOutput[idb][c][idy][idx] * offset_unit;

        grad = (alpha - 1) * c_tl - alpha * c_tr + (1 - alpha) * c_bl + alpha * c_br;
        grad_offset_v += kernels[idb][idc][idy][idx] * grad * gradOutput[idb][c][idy][idx] * offset_unit;
    }

    gradInput_kernels[idb][idc][idy][idx] = grad_kernels;

    gradInput_offsets_h[idb][idc][idy][idx] = grad_offset_h;
    gradInput_offsets_v[idb][idc][idy][idx] = grad_offset_v;
}

void adaptive_gridsampler_kernel_backward(
    const at::Tensor &img,
    const at::Tensor &kernels,
    const at::Tensor &offsets_h,
    const at::Tensor &offsets_v,
    const int *offset_unit,
    const int *padding,
    const at::Tensor &gradOutput,
    at::Tensor *grad_k,
    at::Tensor *gradInput_offsets_h,
    at::Tensor *gradInput_offsets_v) {

    kernel_adaptive_gridsampler_backward<float><<<((*grad_k).numel() + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0>>>(
        img.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        kernels.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offsets_h.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offsets_v.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        offset_unit,
        padding,
        gradOutput.packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        (*grad_k).packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        (*gradInput_offsets_h).packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        (*gradInput_offsets_v).packed_accessor32<float, 4, at::RestrictPtrTraits>(),
        (*grad_k).numel());

    checkCudaErrors(hipGetLastError());
}
